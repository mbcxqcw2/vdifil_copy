#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <fstream>
#include <string>
#include <utility>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

#include "filterbank.hpp"
#include "errors.hpp"

using std::cerr;
using std::cout;
using std::endl;
using std::ifstream;
using std::ofstream;
using std::string;
using std::vector;

#define DEBUG 0
#define GPURUN 1
#define NACCUMULATE 4000
#define NPOL 2
#define PERBLOCK 625
#define TIMEAVG 16 
#define TIMESCALE 0.125
#define UNPACKFACTOR 4
#define VDIFSIZE 8000
#define FFTOUT 257
#define FFTUSE 256

struct FrameInfo {
    unsigned int frameno;
    unsigned int refsecond;
    unsigned int refepoch;
};

struct Timing {
    float readtime;
    float scaletime;
    float filtime;
    float savetime;
    float totaltime;
    float intertime;
};

struct FactorFunctor {
    __host__ __device__ float operator()(float val) {
        return val != 0 ? 1.0f/val : val;
    }
};

__constant__ unsigned char kMask[] = {0x03, 0x0C, 0x30, 0xC0};

__global__ void UnpackKernel(unsigned char **in, float **out, size_t samples) {

    // NOTE: Each thread in the block processes 625 incoming bytes
    int idx = blockIdx.x * blockDim.x * PERBLOCK + threadIdx.x;
    int tmod = threadIdx.x % 4;

    // NOTE: Each thread can store one value
    __shared__ unsigned char incoming[1024];

    int outidx = blockIdx.x * blockDim.x * PERBLOCK * 4;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {
        if (idx < samples) {
            for (int ipol = 0; ipol < 2; ++ipol) {
                incoming[threadIdx.x] = in[ipol][idx];
                __syncthreads();
                int outidx2 = outidx + threadIdx.x;
		for (int ichunk = 0; ichunk < 4; ++ichunk) {
                    int inidx = threadIdx.x / 4 + ichunk * 256;
                    unsigned char inval = incoming[inidx];
                    out[ipol][outidx2] = static_cast<float>(static_cast<short>(((inval & kMask[tmod]) >> (2 * tmod))));
                    outidx2 += 1024;
                }
            }
        }
        idx += blockDim.x;
        outidx += blockDim.x * 4;
    }
}

// NOTE: Does not do any frequency averaging
// NOTE: Outputs only the total intensity and no other Stokes parameters
// NOTE: PERBLOCK is the number of output samples per block
__global__ void DetectKernel(hipfftComplex** __restrict__ in, float* __restrict__ out) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        out[outidx] = outvalue;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void DetectScaleKernel(hipfftComplex** __restrict__ in, unsigned char* __restrict__ out, float* __restrict__ means, float* __restrict__ stdevs) {
    int outidx = blockIdx.x * PERBLOCK * FFTUSE + FFTUSE - threadIdx.x - 1;
    int inidx = blockIdx.x * PERBLOCK * TIMEAVG * FFTOUT + threadIdx.x + 1;

    float outvalue = 0.0f;
    hipfftComplex polval;

    int scaled = 0;

    for (int isamp = 0; isamp < PERBLOCK; ++isamp) {

        // NOTE: Read the data from the incoming array
        for (int ipol = 0; ipol < 2; ++ipol) {
            for (int iavg = 0; iavg < TIMEAVG; ++iavg) {
                polval = in[ipol][inidx + iavg * FFTOUT];
                outvalue += polval.x * polval.x + polval.y * polval.y;
            }

        }
        outvalue *= TIMESCALE;
        scaled = __float2int_ru((outvalue - means[FFTUSE - threadIdx.x - 1]) / stdevs[FFTUSE - threadIdx.x - 1] * 32.0f + 128.0f);
        if (scaled > 255) {
            scaled = 255;
        } else if (scaled < 0) {
            scaled = 0;
        }
        out[outidx] = (unsigned char)scaled;
        inidx += FFTOUT * TIMEAVG;
        outidx += FFTUSE;
        outvalue = 0.0;
    }
}

__global__ void InitDivFactors(float *factors, size_t togenerate) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    // NOTE: I don't want to be dividing by 0
    // NOTE: idx of 0 will not be used anyway
    if (idx < togenerate) {
        if (idx != 0) {
            factors[idx] = 1.0f / idx;
        } else {
            factors[idx] = idx;
        }
    }
}

__global__ void GetScalingFactorsKernel(float* __restrict__ indata, float *base, float *stdev, float *factors, int processed) {

    // NOTE: Filterbank file format coming in
    //float mean = indata[threadIdx.x];
    float mean = 0.0f;
    // NOTE: Depending whether I save STD or VAR at the end of every run
    // float estd = stdev[threadIdx.x];
    float estd = stdev[threadIdx.x] * stdev[threadIdx.x] * (processed - 1.0f);
    float oldmean = base[threadIdx.x];

    //float estd = 0.0f;
    //float oldmean = 0.0;

    float val = 0.0f;
    float diff = 0.0;
    // NOTE: There are 15625 output time samples per NACCUMULATE frames
    for (int isamp = 0; isamp < 15625; ++isamp) {
        val = indata[isamp * FFTUSE + threadIdx.x];
        diff = val - oldmean;
        mean = oldmean + diff * factors[processed + isamp + 1];
        estd += diff * (val - mean);
        oldmean = mean;
    }
    base[threadIdx.x] = mean;
    stdev[threadIdx.x] = sqrtf(estd / (float)(processed + 15625 - 1.0f));
    // stdev[threadIdx.x] = estd;
}

int main(int argc, char *argv[]) {

    string inpola, inpolb, outfil, config;
    double readsec; 
    bool scaling = false;
    bool saveinter = false;

    if ((argc < 5) || (argv[1] == "-h") || (argv[1] == "--help")) {
        cout << "Incorrect number of arguments!" << endl;
        cout << "Command line options:" << endl
                << "-a <filename> - input file for polarisation a" << endl
                << "-b <filename> - input file for polarisation b" << endl
                << "-o <filename> - output filterbank file" << endl
                << "-c <filename> - input configuration file" << endl
                << "-r <number> - number of seconds to process - CURRENTLY NOT WORKING" << endl
                << "-s - enable scaling from 32 bits to 8 bits" << endl
                << "-i - enable saving the intermediate data products" << endl
                << "-h, --help - display this message" << endl;
        exit(EXIT_SUCCESS);
    }

    for (int iarg = 0; iarg < argc; ++iarg) {
        if (std::string(argv[iarg]) == "-a") {
            iarg++;
            inpola = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-b") {
            iarg++;
            inpolb = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-o") {
            iarg++;
            outfil = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-c") {
            iarg++;
            config = std::string(argv[iarg]);
        } else if (std::string(argv[iarg]) == "-s") {
            cout << "Will scale the data to 8 bits" << endl;
            scaling = true;
        } else if (std::string(argv[iarg]) == "-i") {
            cout << "Will save the intermediate products" << endl;
            saveinter = true;
        } else if (std::string(argv[iarg]) == "-r") {
            iarg++;
            readsec = std::stod(argv[iarg]);
        }
    }

    cout << "Input files: " << inpola << " " << inpolb << endl;

    FilHead filhead;
    ReadFilterbankHeader(config, filhead);
    
    if (scaling) {
        filhead.nbits = 8;
    }

    // TODO: This will be wrong for R2C FFT
    filhead.tsamp = 1.0 / (2.0 * filhead.foff) * 2 * FFTUSE * TIMEAVG;
    // TODO: Make sure it is the middle of the top frequency channel
    filhead.fch1 = (filhead.fch1 + filhead.foff / 2.0f) * 1e-06;
    filhead.nchans = FFTUSE;
    filhead.foff = -1.0 * filhead.foff / FFTUSE * 1e-06 ;

    filhead.fch1 = filhead.fch1 + filhead.foff / 2.0;

    if (DEBUG) {
        cout << "Some header info:\n"
                << "Raw file: " << filhead.rawfile << endl
                << "Source name: " << filhead.source << endl
                << "Azimuth: " << filhead.az << endl
                << "Zenith angle: " << filhead.za << endl
                << "Declination: " << filhead.dec << endl
                << "Right ascension: " << filhead.ra << endl
                << "Top channel frequency: " << filhead.fch1 << endl
                << "Channel bandwidth: " << filhead.foff << endl
                << "Number of channels: " << filhead.nchans << endl
                << "Sampling time: " << filhead.tsamp << endl
                << "Bits per sample: " << filhead.nbits << endl;
    }

    // TODO: Make sure there are correct values for bandwidth and sampling time in the header after taking averaging into account

    ifstream filepola(inpola.c_str(), ifstream::in | ifstream::binary);
    ifstream filepolb(inpolb.c_str(), ifstream::in | ifstream::binary);
    ofstream filfile(outfil.c_str(), ofstream::out | ofstream::binary);

    if (!filepola || !filepolb) {
	if (!filepola) {
            cout << "Could not open file " << inpola << endl;
        }
        if (!filepolb) {
            cout << "Could not open file " << inpolb << endl;
        }
        exit(EXIT_FAILURE);
    }
    // TODO: Can save the filterbank header straight away, after the first header is read
    unsigned char vdifheadpola[32];
    unsigned char vdifheadpolb[32];
    filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
    filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);

    filepola.seekg(0, filepola.end);
    long long filelengtha = filepola.tellg();
    filepola.seekg(0, filepola.beg);

    filepolb.seekg(0, filepolb.end);
    long long filelengthb = filepolb.tellg();
    filepolb.seekg(0, filepolb.beg);

    unsigned int startframe;
    unsigned int startsecond;

    cout << filelengtha << " " << filelengthb << endl;

    startframe = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));	// frame number in this second
    startsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));

    if (DEBUG) {
        cout << "Starting time: " << startsecond << ":" << startframe << endl;
    }

    // NOTE: Need to read headers in
    unsigned int toread = NACCUMULATE * 8000;
    // NOTE: No more headers after unpacking
    unsigned int unpackedsize = NACCUMULATE * VDIFSIZE * UNPACKFACTOR;
    unsigned int fftedsize = unpackedsize / (2 * FFTUSE) * FFTOUT;
    unsigned int powersize = unpackedsize / (2 * FFTUSE) * FFTUSE / TIMEAVG;

    hipfftHandle fftplan;
    int fftsizes[1];
    fftsizes[0] = 2 * FFTUSE;
    int fftbatchsize = unpackedsize / fftsizes[0];
    cout << fftbatchsize << endl;
    cufftCheckError(hipfftPlanMany(&fftplan, 1, fftsizes, NULL, 1, FFTUSE, NULL, 1, FFTUSE, HIPFFT_R2C, fftbatchsize));

    unsigned char *tmppola = new unsigned char[toread];
    unsigned char *tmppolb = new unsigned char[toread];

    unsigned char *devpola;
    unsigned char *devpolb;
    unsigned char **datapol = new unsigned char*[NPOL];
    unsigned char **devpol;
    float **unpacked = new float*[NPOL];
    float **devunpacked;
    hipfftComplex **ffted = new hipfftComplex*[NPOL];
    hipfftComplex **devffted;
    
    unsigned char *devpower;
    unsigned char *tmppower = new unsigned char[powersize * filhead.nbits / 8];

    if (GPURUN) {
        cudaCheckError(hipMalloc((void**)&devpola, toread * sizeof(unsigned char)));
        cudaCheckError(hipMalloc((void**)&devpolb, toread * sizeof(unsigned char)));

        cudaCheckError(hipMalloc((void**)&devpol, NPOL * sizeof(unsigned char*)));
        cudaCheckError(hipMalloc((void**)&datapol[0], toread * sizeof(unsigned char)));
        cudaCheckError(hipMalloc((void**)&datapol[1], toread * sizeof(unsigned char)));
        cudaCheckError(hipMemcpy(devpol, datapol, NPOL * sizeof(unsigned char*), hipMemcpyHostToDevice));

        cudaCheckError(hipMalloc((void**)&devunpacked, NPOL * sizeof(float*)));
        cudaCheckError(hipMalloc((void**)&unpacked[0], unpackedsize * sizeof(float)));
        cudaCheckError(hipMalloc((void**)&unpacked[1], unpackedsize * sizeof(float)));
        cudaCheckError(hipMemcpy(devunpacked, unpacked, NPOL * sizeof(float*), hipMemcpyHostToDevice));

        cudaCheckError(hipMalloc((void**)&devffted, NPOL * sizeof(hipfftComplex*)));
        cudaCheckError(hipMalloc((void**)&ffted[0], fftedsize * sizeof(hipfftComplex)));
        cudaCheckError(hipMalloc((void**)&ffted[1], fftedsize * sizeof(hipfftComplex)));
        cudaCheckError(hipMemcpy(devffted, ffted, NPOL * sizeof(hipfftComplex*), hipMemcpyHostToDevice));

        cudaCheckError(hipMalloc((void**)&devpower, powersize * (filhead.nbits / 8)));
    }

    vector<std::pair<FrameInfo, FrameInfo>> vdifframes;

    FrameInfo tmpframea, tmpframeb;
    int refsecond;
    int frameno;
    int epoch;

    WriteFilterbankHeader(filfile, filhead);
   
    Timing runtimes;
    runtimes.readtime = 0.0f;
    runtimes.scaletime = 0.0f;
    runtimes.filtime = 0.0f;
    runtimes.savetime = 0.0f;
    runtimes.totaltime = 0.0f;
    runtimes.intertime = 0.0f;

    std::chrono::time_point<std::chrono::steady_clock> readstart, readend, scalestart, scaleend, filstart, filend, savestart, saveend, interstart, interend;

    float *tmpunpackeda = new float[unpackedsize];
    float *tmpunpackedb = new float[unpackedsize];
    hipfftComplex *tmpffta = new hipfftComplex[fftedsize];
    hipfftComplex *tmpfftb = new hipfftComplex[fftedsize];

    bool saved = false;

    //float *dmeans;
    //float *dstdevs;
    //cudaCheckError(hipMalloc((void**)&dmeans, FFTUSE * sizeof(float)));
    //cudaCheckError(hipMalloc((void**)&dstdevs, FFTUSE * sizeof(float)));

    thrust::device_vector<float> dmeans, dstdevs;
    dmeans.resize(FFTUSE);
    dstdevs.resize(FFTUSE);
    thrust::fill(dmeans.begin(), dmeans.end(), 0.0f);
    thrust::fill(dstdevs.begin(), dstdevs.end(), 0.0f);
    float *pdmeans = thrust::raw_pointer_cast(dmeans.data());
    float *pdstdevs = thrust::raw_pointer_cast(dstdevs.data());    

    cout << "Size of the device vectors: " << dmeans.size() << " " << dstdevs.size() << endl;

    scalestart = std::chrono::steady_clock::now();

    // NOTE: Use first 5 accumulates of data to obtain scaling factors
    if (scaling) {

        size_t divfactors = 5 * powersize / FFTUSE;
        thrust::device_vector<float> dfactors; 
        dfactors.resize(divfactors + 1);
        thrust::sequence(dfactors.begin(), dfactors.end());
        thrust::transform(dfactors.begin(), dfactors.end(), dfactors.begin(), FactorFunctor());
        float *pdfactors = thrust::raw_pointer_cast(dfactors.data());

        //float *dfactors;
        //size_t divfactors = 5 * powersize / FFTUSE;
        //cudaCheckError(hipMalloc((void**)&dfactors, divfactors * sizeof(float)));
        //int scalethreads = 1024;
        //int scaleblocks = (divfactors - 1) / scalethreads + 1;
        //cout << "Div factors blocks: " << scaleblocks << " and threads: " << scalethreads << endl;
        //InitDivFactors<<<scaleblocks, scalethreads>>>(dfactors, divfactors);
        //cudaCheckError(hipDeviceSynchronize());
        //cudaCheckError(hipGetLastError());
        size_t processed = 0;

        float *tmpdpower;
        cudaCheckError(hipMalloc((void**)&tmpdpower, powersize * sizeof(float)));

	while((filepola.tellg() < (5 * NACCUMULATE * 8032)) && (filepolb.tellg() < (5 * NACCUMULATE * 8032))) {
            for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
                filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
                filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
                filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
                filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
            }

            cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
            cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));

            UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
            for (int ipol = 0; ipol < NPOL; ++ipol) {
                cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
            }
            DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, tmpdpower);
            cudaCheckError(hipDeviceSynchronize());
            GetScalingFactorsKernel<<<1, FFTUSE, 0, 0>>>(tmpdpower, pdmeans, pdstdevs, pdfactors, processed);
            processed += (powersize / FFTUSE);
            cudaCheckError(hipDeviceSynchronize());
        }

        //float *hmeans = new float[FFTUSE];
        //float *hstdevs = new float[FFTUSE];

        //cudaCheckError(hipMemcpy(hmeans, dmeans, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));
        //cudaCheckError(hipMemcpy(hstdevs, dstdevs, FFTUSE * sizeof(float), hipMemcpyDeviceToHost));

        thrust::host_vector<float> hmeans = dmeans;
        thrust::host_vector<float> hstdevs = dstdevs;

        std::ofstream statsfile("mean_stdev.dat");

        cout << "Size of host vector:" << hmeans.size() << endl;
 
        if (statsfile) {
            for (int ichan = 0; ichan < hmeans.size(); ++ichan) {
                statsfile << hmeans[ichan] << " " << hstdevs[ichan] << endl;
            }
        } else {
            cerr << "Could not open the stats file" << endl;
        }

        statsfile.close();

        hipFree(tmpdpower);
         
    }

    scaleend = std::chrono::steady_clock::now();

    runtimes.scaletime = std::chrono::duration<float>(scaleend - scalestart).count();

    filepola.seekg(0, filepola.beg);
    filepolb.seekg(0, filepolb.beg);

    std::ofstream unpackedfilea ((outfil + ".unp0").c_str(), std::ios_base::binary);
    std::ofstream unpackedfileb ((outfil + ".unp1").c_str(), std::ios_base::binary);
    std::ofstream fftfilea ((outfil + ".fft0").c_str(), std::ios_base::binary);
    std::ofstream fftfileb ((outfil + ".fft1").c_str(), std::ios_base::binary);

    while((filepola.tellg() < (filelengtha - NACCUMULATE * 8000)) && (filepolb.tellg() < (filelengthb - NACCUMULATE * 8000))) {
        //cout << filepola.tellg() << endl;
        // NOTE: This implementation
        for (int iacc = 0; iacc < NACCUMULATE; ++iacc) {
	    readstart = std::chrono::steady_clock::now();
            filepola.read(reinterpret_cast<char*>(vdifheadpola), 32);
            filepolb.read(reinterpret_cast<char*>(vdifheadpolb), 32);
            filepola.read(reinterpret_cast<char*>(tmppola) + iacc * 8000, 8000);
            filepolb.read(reinterpret_cast<char*>(tmppolb) + iacc * 8000, 8000);
            readend = std::chrono::steady_clock::now();
            runtimes.readtime += std::chrono::duration<float>(readend - readstart).count();

            refsecond = (unsigned int)(vdifheadpola[0] | (vdifheadpola[1] << 8) | (vdifheadpola[2] << 16) | ((vdifheadpola[3] & 0x3f) << 24));
            frameno = (unsigned int)(vdifheadpola[4] | (vdifheadpola[5] << 8) | (vdifheadpola[6] << 16));
            epoch = (unsigned int)(vdifheadpola[7] & 0x3f);
//            frameno += (refsecond - startsecond) * 4000;

            tmpframea.frameno = frameno;
            tmpframea.refsecond = refsecond;
            tmpframea.refepoch = epoch;

            refsecond = (unsigned int)(vdifheadpolb[0] | (vdifheadpolb[1] << 8) | (vdifheadpolb[2] << 16) | ((vdifheadpolb[3] & 0x3f) << 24));
            frameno = (unsigned int)(vdifheadpolb[4] | (vdifheadpolb[5] << 8) | (vdifheadpolb[6] << 16));
            epoch = (unsigned int)(vdifheadpolb[7] & 0x3f);
//            frameno += (refsecond - startsecond) * 4000;

            tmpframeb.frameno = frameno;
            tmpframeb.refsecond = refsecond;
            tmpframeb.refepoch = epoch;

            vdifframes.push_back(std::make_pair(tmpframea, tmpframeb));

            // NOTE: Can use subtract startframe to put frame count at 0 and use that to save into the buffer

        }
 
       if (GPURUN) {
            filstart = std::chrono::steady_clock::now();
            cudaCheckError(hipMemcpy(datapol[0], tmppola, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));
            cudaCheckError(hipMemcpy(datapol[1], tmppolb, NACCUMULATE * 8000 * sizeof(unsigned char), hipMemcpyHostToDevice));

            UnpackKernel<<<50, 1024, 0, 0>>>(devpol, devunpacked, toread);
            for (int ipol = 0; ipol < NPOL; ++ipol) {
                cufftCheckError(hipfftExecR2C(fftplan, unpacked[ipol], ffted[ipol]));
            }

            if (filhead.nbits == 8) {
                DetectScaleKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<unsigned char*>(devpower), pdmeans, pdstdevs);
            } else if (filhead.nbits == 32) {
                DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
            } else {
                cerr << "Unsupported option! Will use float!" << endl;
                DetectKernel<<<25, FFTUSE, 0, 0>>>(devffted, reinterpret_cast<float*>(devpower));
            }

            //PowerKernel<<<25, FFTUSE, 0, 0>>>(devffted, devpower);
            cudaCheckError(hipDeviceSynchronize());
            cudaCheckError(hipMemcpy(tmppower, devpower, powersize * filhead.nbits / 8, hipMemcpyDeviceToHost));
            
            if (!saved) {
                std::ofstream unpackedfile("unpacked.dat");
		cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
		cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], 2 * 8000 * 4 * sizeof(float), hipMemcpyDeviceToHost));
		for (int isamp = 0; isamp < 2 * 8000 * 4; ++isamp) {
                    unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
                }
                unpackedfile.close();
                saved = true;
            }

            filend = std::chrono::steady_clock::now();
            runtimes.filtime += std::chrono::duration<float>(filend - filstart).count();
            
            savestart = std::chrono::steady_clock::now(); 
            filfile.write(reinterpret_cast<char*>(tmppower), powersize * filhead.nbits / 8);
            saveend = std::chrono::steady_clock::now();
            runtimes.savetime += std::chrono::duration<float>(saveend - savestart).count();

   

            if (saveinter) {

                interstart = std::chrono::steady_clock::now();

                cudaCheckError(hipMemcpy(tmpunpackeda, unpacked[0], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
		cudaCheckError(hipMemcpy(tmpunpackedb, unpacked[1], unpackedsize * sizeof(float), hipMemcpyDeviceToHost));
                /*for (int isamp = 0; isamp < unpackedsize; ++isamp) {
                    unpackedfile << tmpunpackeda[isamp] << " " << tmpunpackedb[isamp] << endl;
                }*/

                unpackedfilea.write(reinterpret_cast<char*>(tmpunpackeda), unpackedsize * sizeof(float));
                unpackedfileb.write(reinterpret_cast<char*>(tmpunpackedb), unpackedsize * sizeof(float));

                cudaCheckError(hipMemcpy(tmpffta, ffted[0], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
                cudaCheckError(hipMemcpy(tmpfftb, ffted[1], fftedsize * sizeof(hipfftComplex), hipMemcpyDeviceToHost));
                /*for (int isamp = 0; isamp < fftedsize; ++isamp) {
                    fftfile << tmpffta[isamp].x << " " << tmpffta[isamp].y << " " << tmpfftb[isamp].x << " " << tmpfftb[isamp].y << endl;
                }*/

                fftfilea.write(reinterpret_cast<char*>(tmpffta), fftedsize * sizeof(hipfftComplex));
                fftfileb.write(reinterpret_cast<char*>(tmpfftb), fftedsize * sizeof(hipfftComplex));

                interend = std::chrono::steady_clock::now();
                runtimes.intertime += std::chrono::duration<float>(interend - interstart).count();           

            }

        }
        cout << "Completed " << std::fixed << std::setprecision(2) << (float)filepola.tellg() / (float)(filelengtha - 1.0) * 100.0f << "%\r";
        cout.flush();
    }

    cout << endl;
    filfile.close();
    unpackedfilea.close();
    unpackedfileb.close();
    fftfilea.close();
    fftfileb.close();

    runtimes.totaltime = runtimes.readtime + runtimes.scaletime + runtimes.filtime + runtimes.savetime + runtimes.intertime;

    cout << "Total execution time: " << runtimes.totaltime << "s\n";
    cout << "\tScaling factors: " << runtimes.scaletime << "s\n";
    cout << "\tFile read: " << runtimes.readtime << "s\n";
    cout << "\tFilterbanking: " << runtimes.filtime << "s\n";
    cout << "\tFile write: " << runtimes.savetime << "s\n";
    if (saveinter) {
        cout << "\tIntermediate write: " << runtimes.intertime << "s\n";
    }

    if (DEBUG) {
        std::ofstream outframes("dataframes.dat");

        outframes << "Ref Epoch A\tRef second A\tRef frame A\tRef Epoch B\tRef second B\tRef frame b\n";
        for (auto iframe = vdifframes.begin(); iframe != vdifframes.end(); ++iframe) {
            outframes << iframe->first.refepoch << "\t" << iframe->first.refsecond << "\t" << iframe->first.frameno << "\t"
            << iframe->second.refepoch << "\t" << iframe->second.refsecond << "\t" << iframe->second.frameno << endl;
        }

        outframes.close();
    }
    return 0;
}
